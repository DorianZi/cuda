#include <iostream>
#include <unistd.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
using namespace std;

__global__ void infinitekernel(float *dptr, int *dwait)
{
 	while(*dwait)	*dptr += 1;
	*dptr = 999;

}	

int main(void)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipStream_t stream[2];
	for (int i=0; i < 2 ; i++)
		hipStreamCreate(&stream[i]);
	float *hptr;
	float *dptr;
	int *hwait;
	int *dwait;
	hptr = (float*)malloc(sizeof(float));
	hwait = (int*)malloc(sizeof(int));
	hipMalloc((void **)&dptr, sizeof(float));
	hipMalloc((void **)&dwait, sizeof(int));
	*hptr = 9;
	*hwait = 1;
	hipMemcpyAsync(dptr, hptr, sizeof(float), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(dwait, hwait, sizeof(float), hipMemcpyHostToDevice, stream[0]);
        
	cout << "Start to record kernel elapsed time" << endl;
	hipEventRecord(start, stream[1]);
	infinitekernel<<<1, 1, 0, stream[1]>>>(dptr,dwait);
        hipEventRecord(stop, stream[1]);
	
	for(int i=0; i<5; i++)
	{
		sleep(1);
		hipMemcpyAsync(hptr, dptr, sizeof(float), hipMemcpyDeviceToHost, stream[0]);
	        cout << "["<< i << " seconds]" <<"value = " << *hptr << endl;
	}

	*hwait = 0;
	hipMemcpyAsync(dwait, hwait, sizeof(int), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(hptr, dptr, sizeof(float), hipMemcpyDeviceToHost, stream[0]);
	cout <<"[Finally]" << "value = "<< *hptr << endl;
	
	hipEventSynchronize(stop);
	float elapsTime;
	hipEventElapsedTime(&elapsTime, start, stop);
	cout << "Elapsed Time: " << elapsTime << endl;
}
