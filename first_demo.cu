#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 9

__global__ void kernel(int *ptr)
{
	*ptr = *ptr + N;
}

int main(void)
{
    int computeMajor;
    int computeMinor;
    hipDeviceGetAttribute(&computeMajor, hipDeviceAttributeComputeCapabilityMajor,0);
    hipDeviceGetAttribute(&computeMinor, hipDeviceAttributeComputeCapabilityMinor,0);
    printf("Compute Capability: %d.%d\n", computeMajor, computeMinor);
    int *hptr;
    int *dptr;
    size_t size = sizeof(int);
    hptr = (int *)malloc(size);
    hipMalloc((void **)&dptr, size);
    //memset(hptr, 1, 1);
    *hptr = 1;
    printf("%d + %d = ", *hptr, N);
    hipMemcpy(dptr, hptr, size, hipMemcpyHostToDevice);
    kernel<<<2,3>>>(dptr); 
    hipMemcpy(hptr, dptr, size, hipMemcpyDeviceToHost);
    printf("%d\n", *hptr);
    free(hptr);
    hipFree(dptr);

}
