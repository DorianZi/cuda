#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 9

__global__ void kernel(int *ptr)
{
	*ptr = *ptr + N;
}

int main(void)
{
    int *hptr;
    int *dptr;
    size_t size = sizeof(int);
    hptr = (int *)malloc(size);
    hipMalloc((void **)&dptr, size);
    //memset(hptr, 1, 1);
    *hptr = 1;
    printf("%d + %d = ", *hptr, N);
    hipMemcpy(dptr, hptr, size, hipMemcpyHostToDevice);
    kernel<<<2,3>>>(dptr); 
    hipMemcpy(hptr, dptr, size, hipMemcpyDeviceToHost);
    printf("%d\n", *hptr);
    free(hptr);
    hipFree(dptr);

}
