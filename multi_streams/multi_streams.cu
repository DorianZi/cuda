#include <iostream>
#include <unistd.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
using namespace std;

__global__ void infinitekernel(float *dptr, int *dwait)
{
 	while(*dwait)	*dptr += 1;
	*dptr = 999;

}	

int main(void)
{
	hipStream_t stream[2];
	for (int i=0; i < 2 ; i++)
		hipStreamCreate(&stream[i]);
	float *hptr;
	float *dptr;
	int *hwait;
	int *dwait;
	hptr = (float*)malloc(sizeof(float));
	hwait = (int*)malloc(sizeof(int));
	hipMalloc((void **)&dptr, sizeof(float));
	hipMalloc((void **)&dwait, sizeof(int));
	*hptr = 9;
	*hwait = 1;
	hipMemcpyAsync(dptr, hptr, sizeof(float), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(dwait, hwait, sizeof(float), hipMemcpyHostToDevice, stream[0]);
	infinitekernel<<<1, 1, 0, stream[1]>>>(dptr,dwait);

	for(int i=0; i<10; i++)
	{
		sleep(1);
		hipMemcpyAsync(hptr, dptr, sizeof(float), hipMemcpyDeviceToHost, stream[0]);
	        cout << "["<< i << " seconds]" <<"value = " << *hptr << endl;
	}

	*hwait = 0;
	hipMemcpyAsync(dwait, hwait, sizeof(int), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(hptr, dptr, sizeof(float), hipMemcpyDeviceToHost, stream[0]);
	cout <<"[Finally]" << "value = "<< *hptr << endl;
}
